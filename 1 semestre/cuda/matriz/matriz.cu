// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)

#include <hip/hip_runtime.h>
#include <stdio.h>

typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C){
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char* argv[]){
  Matrix A, B, C;
  int a1, a2, b1, b2;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Read some values from the commandline
  a1 = atoi(argv[1]);           /* Height of A */
  a2 = atoi(argv[2]);           /* Width  of A */
  b1 = a2;                  /* Height of B */
  b2 = atoi(argv[3]);           /* Width  of B */

  A.height = a1;
  A.width = a2;
  A.elements = (float*)malloc(A.width * A.height * sizeof(float));

  B.height = b1;
  B.width = b2;
  B.elements = (float*)malloc(B.width * B.height * sizeof(float));

  C.height = A.height;
  C.width = B.width;
  C.elements = (float*)malloc(C.width * C.height * sizeof(float));

  for(int i = 0; i < A.height; i++)
    for(int j = 0; j < A.width; j++)
      A.elements[i*A.width + j] = (float)(rand() % 3);

  for(int i = 0; i < B.height; i++)
    for(int j = 0; j < B.width; j++)
      B.elements[i*B.width + j] = (float)(rand() % 2);

  hipEventRecord(start);
  MatMul(A, B, C);
  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("%d x %d Bloque:%d -- time: %f\n",a1,a2,BLOCK_SIZE,milliseconds);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}